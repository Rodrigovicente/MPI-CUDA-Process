#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <mpi.h>

#define NUM_THREADS 512

#define min(a, b) (((a) < (b)) ? (a) : (b))

__device__ void game_of_life(int *univ, int w, int size, int id, int *new_univ) {

	// Neighbor positions
	unsigned int x = id % w;
	unsigned int y = id - x;
	unsigned int x_l = x - 1;
	unsigned int x_r = x + 1;
	unsigned int y_u = y - w;
	unsigned int y_d = y + w;

	int n_alive;

	// Calculate number of alive neighbors
	if (y_u < 0) {
		if (x_l < 0) {
			n_alive = univ[x_r + y] + univ[x + y_d] + univ[x_r + y_d];

		}
		else if (x_r > w) {
			n_alive = univ[x_l + y] + univ[x_l + y_d] + univ[x + y_d];

		}
		else {
			n_alive = univ[x_l + y] + univ[x_r + y] + univ[x_l + y_d] + univ[x + y_d] + univ[x_r + y_d];

		}
	}
	else if (y_d < 0) {
		if (x_l < 0) {
			n_alive = univ[x + y_u] + univ[x_r + y_u] + univ[x_r + y];

		}
		else if (x_r > w) {
			n_alive = univ[x_l + y_u] + univ[x + y_u] + univ[x_l + y];

		}
		else {
			n_alive = univ[x_l + y_u] + univ[x + y_u] + univ[x_r + y_u] + univ[x_l + y] + univ[x_r + y];

		}
	}
	else {
		if (x_l < 0) {
			n_alive = univ[x + y_u] + univ[x_r + y_u] + univ[x_r + y] + univ[x + y_d] + univ[x_r + y_d];

		}
		else if (x_r > w) {
			n_alive = univ[x_l + y_u] + univ[x + y_u] + univ[x_l + y] + univ[x_l + y_d] + univ[x + y_d];

		}
		else {
			n_alive = univ[x_l + y_u] + univ[x + y_u] + univ[x_r + y_u] + univ[x_l + y] + univ[x_r + y] + univ[x_l + y_d] + univ[x + y_d] + univ[x_r + y_d];

		}
	}

	new_univ[x + y] = n_alive == 3 || (n_alive == 2 && univ[x + y]) ? 1 : 0;

}

__global__ void middle_kernel(int *univ, int h, int w, int p_id, int *new_univ) {
	int id = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int size = h * w;
	//printf("%d %d %d -> %d\n", blockIdx.x, threadIdx.x, blockDim.x, id);

	if (p_id == 0) {
		if (id < (size / 2) - w) { // Caso n�o seja borda compartilhada
			game_of_life(univ, w, size, id, new_univ);

		}
	} else if (p_id == 1) {
		if ((id >= (size / 2) + w) && (id <= size)) { // Caso n�o seja borda compartilhada
			game_of_life(univ, w, size, id, new_univ);

		}
		else {
			new_univ[id] = 0;
		}
	}
}

__global__ void border_kernel(int *univ, int h, int w, int p_id, int *new_univ) {
	int id = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int size = h * w;
	//printf("%d %d %d -> %d\n", blockIdx.x, threadIdx.x, blockDim.x, id);
	
	if (p_id == 0) {
		if ((id >= (size / 2) - w) && (id < size / 2)) { // Caso SEJA borda compartilhada
			game_of_life(univ, w, size, id, new_univ);

		} 
		else {
			new_univ[id] = 0;
		}
	} else if (p_id == 1) {
		if ((id >= size / 2) && (id < (size) / 2 + w)) { // Caso SEJA borda compartilhada
			game_of_life(univ, w, size, id, new_univ);

		}
		else {
			new_univ[id] = 0;
		}
	}
}

void print_array(int arr[], int w, int size) {
	printf("\n");

	for (int i = 0; i < size; i++)
	{
		printf("%s", (arr[i] == 1 ? "0" : " "));
		//printf("%d", arr[i]);

		if ((i + 1) % w == 0) {
			printf("\n");
		}
	}

	printf("\n");
}

void create_universe(int *univ, int h, int w, float prob) {
	float rand_prob;

	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			int k = (i * w) + j;
			if (i == 0 || j == 0 || i == h - 1 || j == w - 1) {
				univ[k] = 0;
			}
			else {
				rand_prob = (float)rand() / RAND_MAX;
				univ[k] = rand_prob > prob ? 0 : 1;
			}
		}
	}
}

int main(int argc, char **argv)
{
	int g, h, w;

	printf("Enter desired number of generations:\n");
	scanf("%d", &g);

	printf("Enter desired height of universe:\n");
	scanf("%d", &h);

	printf("Enter desired width of universe:\n");
	scanf("%d", &w);

	hipStream_t border_p1_stream;
	hipStream_t middle_p1_stream;
	hipStream_t border_p2_stream;
	hipStream_t middle_p2_stream;
	hipStreamCreate(&border_p1_stream);
	hipStreamCreate(&middle_p1_stream);
	hipStreamCreate(&border_p2_stream);
	hipStreamCreate(&middle_p2_stream);


	MPI_Status status;
	int p_id, p_group, p_name;
	char processor_name[MPI_MAX_PROCESSOR_NAME];

	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &p_group);
	MPI_Comm_rank(MPI_COMM_WORLD, &p_id);
	MPI_Get_processor_name(processor_name, &p_name);

	// Number of cells in universe
	int size = h * w;

	// Host(CPU) arrays
	int *h_univ = (int*)malloc(size * sizeof(int));
	int *h_new_univ = (int*)malloc(size * sizeof(int));

	// Devide(GPU) arrays
	int *d_univ;
	int *d_new_univ;
	hipMalloc((void**)&d_univ, size * sizeof(int));
	hipMalloc((void**)&d_new_univ, size * sizeof(int));

	create_universe(h_univ, h, w, 0.15);

	size_t n_threads = size > NUM_THREADS ? NUM_THREADS : size;
	unsigned n_blocks = size > NUM_THREADS ? (unsigned)size / NUM_THREADS : (unsigned)1;
	//printf("size: %d - blocks: %d - threads: %d\n", size, n_blocks, t);

	int my_part;
	int iter_count = g;

	if (p_id == 0) {
		
		while (iter_count > 0) {

			my_part = (h * w) / 2;

			hipMemcpyAsync(d_univ, h_univ, size * sizeof(int), hipMemcpyHostToDevice, middle_p1_stream); // passa matriz para a GPU
			middle_kernel <<<n_blocks, n_threads, 0, middle_p1_stream >>> (d_univ, h, w, p_id, d_new_univ); // processa a matriz

			//std::swap(d_univ, d_new_univ);

			if (iter_count < g) {
				for (int i = 1; i < w - 1; i++) {
					MPI_Recv(&h_univ[my_part + i], 1, MPI_INT, 1, 1, MPI_COMM_WORLD, &status); // recebe borda do outro processo
				}

				hipMemcpyAsync(d_univ, h_univ, size * sizeof(int), hipMemcpyHostToDevice, border_p1_stream); // passa a matriz para a GPU em outra stream (para processamento paralelo)
			}
			
			border_kernel <<<n_blocks, n_threads, 0, border_p1_stream >>>(d_univ, h, w, p_id, d_new_univ);
			hipDeviceSynchronize();

			hipMemcpyAsync(h_univ, d_univ, size * sizeof(int), hipMemcpyDeviceToHost, border_p1_stream);

			// Envia a borda para o pr�ximo processo
			my_part = my_part - w;
			for (int i = 1; i < w - 1; i++) {
				MPI_Send(&h_univ[my_part + i], 1, MPI_INT, 1, 1, MPI_COMM_WORLD);
			}

			// print_array(h_univ, w, size);
			iter_count--;
		}
	}
	else {
		while (iter_count > 0) {

			my_part = (h * w) / 2;

			hipMemcpyAsync(d_univ, h_univ, size * sizeof(int), hipMemcpyHostToDevice, middle_p2_stream); // passa matriz para a GPU
			middle_kernel << <n_blocks, n_threads, 0, middle_p2_stream >> > (d_univ, h, w, p_id, d_new_univ); // processa a matriz

																											  //std::swap(d_univ, d_new_univ);

			if (iter_count < g) {
				for (int i = 1; i < w - 1; i++) {
					MPI_Recv(&h_univ[my_part + i], 1, MPI_INT, 1, 1, MPI_COMM_WORLD, &status); // recebe borda do outro processo
				}
			}

			hipMemcpyAsync(d_univ, h_univ, size * sizeof(int), hipMemcpyHostToDevice, border_p2_stream); // passa a matriz para a GPU em outra stream (para processamento paralelo)

			border_kernel << <n_blocks, n_threads, 0, border_p2_stream >> >(d_univ, h, w, p_id, d_new_univ);
			hipDeviceSynchronize();

			hipMemcpyAsync(h_univ, d_univ, size * sizeof(int), hipMemcpyDeviceToHost, border_p2_stream);

			// Envia a borda para o pr�ximo processo
			my_part = my_part - w;
			for (int i = 1; i < w - 1; i++) {
				MPI_Send(&h_univ[my_part + i], 1, MPI_INT, 1, 1, MPI_COMM_WORLD);
			}

			// print_array(h_univ, w, size);
			iter_count--;
		}

	}
	// Release memory? 
	free(h_univ);
	free(h_new_univ);
	hipFree(d_univ);
	hipFree(d_new_univ);
	hipStreamDestroy(border_p1_stream);
	hipStreamDestroy(middle_p1_stream);
	hipStreamDestroy(border_p2_stream);
	hipStreamDestroy(middle_p2_stream);


	MPI_Finalize();

	return 0;
}